#include "hip/hip_runtime.h"
//
// PAising version 1.12. This program employs multi-spin coding.
// This program is introduced in the paper:
// L.Yu. Barash, M. Weigel, M. Borovsky, W. Janke, L.N. Shchur, GPU accelerated population annealing algorithm
// This program is licensed under a Creative Commons Attribution 4.0 International License:
// http://creativecommons.org/licenses/by/4.0/
//
// Use command line option -? to print list of available command line options.
// All of the command line options are optional. 
//

#include <iostream>
#include <fstream>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#ifdef _WIN32			// this program is compatible with any of the Windows, Unix/Linux, MacOS environments
	#include <direct.h>
#else
	#include <sys/stat.h>
#endif

// #define MHR			// uncomment/comment to enable/disable multi-histogram reweighting
// #define AdaptiveStep		// uncomment/comment to enable/disable adaptive temperature step
// #define EnergiesPopStore	// uncomment/comment to enable/disable storing energies at each T

#define L	64		// linear size of the system in x,y direction
#define Ldiv2   (L/2)
#define N       (L*L)

#define RNGseed	time(NULL)	// Use 32-bit integer as a seed for random number generation, e.g., time(NULL) 

typedef hiprandStatePhilox4_32_10_t RNGState;

#define MSbits	32		// Use 8, 16, 32 or 64 Multi-spin bits per word

unsigned int EQsweeps = 100;			// number of equilibration sweeps

double Binit = 0;				// initial inverse temperature
double Bfin = 1;				// final inverse temperature
double dBinit = 0.005;				// inverse temperature step

#ifdef AdaptiveStep
	double MinOverlap = 0.85;		// minimal value of acceptable overlap of energy histograms
	double MaxOverlap = 0.87;		// maximal value of acceptable overlap of energy histograms
#endif

int Rinit = 20000;				// Initial size of population of replicas

int runs = 1;					// number of population annealing algorithm independent runs

int OutputPrecision = 11;			// precision (number of digits) of the output

const unsigned int AA = 1664525;		// linear congruential generator parameters
const unsigned int CC = 1013904223;

#ifdef MHR
	const short MHR_Niter = 1;	// number of iterations for multi-histogram analysis (single iteration is usually sufficient)
#endif

const int boltzTableL = 2;			// Boltzmann factor table length
const int nBmax = 10000;			// number of temperature steps should not exceed nBmax

texture<unsigned int,1,hipReadModeElementType> boltzT;
using namespace std;

#define EQthreads 128	// number of threads per block for the equilibration kernel
#define Nthreads  1024	// number of threads per block for the parallel reduction algorithm
// Use Nthreads=1024 for CUDA compute capability 2.0 and above; Nthreads=512 for old devices with CUDA compute capability 1.x.

double* Qd; double* ioverlapd;

#if   MSbits == 8
	#define MultiSpin signed char
#elif MSbits == 16
	#define MultiSpin signed short
#elif MSbits == 32
	#define MultiSpin signed int
#elif MSbits == 64
	#define MultiSpin signed long long int
#endif

// struct Replica covers all information about the replica including its configuration, sublattice magnetizations,
// internal energy and number of replica's offspring
struct Replica{
		MultiSpin gA[N/2];	// sublattice configurations with multipsin-coding = one value in array represents
		MultiSpin gB[N/2];	// spins of 8 different replicas in the same site in lattice
		int IE[MSbits];				// internal energy
		int M[MSbits];				// magnetization
		unsigned int Roff[MSbits];		// number of replica's offspring
		union{double ValDouble[2]; unsigned int ValInt[MSbits+2];} parSum;  // these variables are used for storing sums
		bool isActive[MSbits];			// isActive[i] determines if the i-th replica is active
};

// CUDA error checking macro
#define CUDAErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s ; %s ; line %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

template <class sometype> __inline__ __device__ sometype smallblockReduceSum(sometype val) // use when blockDim.x < 32
{											   // blockDim.x must be a power of 2
	static __shared__ sometype shared[32];
	shared[threadIdx.x] = val;
	for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
		__syncthreads(); if (threadIdx.x < stride)  shared[threadIdx.x] += shared[threadIdx.x+stride];
	}
	__syncthreads(); return shared[0];
}

#if  (defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 300)
template <class sometype> __inline__ __device__ sometype warpReduceSum(sometype val)
{
	for (int offset = warpSize/2; offset > 0; offset /= 2) val += __shfl_down(val, offset);
	return val;
}

template <class sometype> __inline__ __device__ sometype blockReduceSum(sometype val)	 // use when blockDim.x is divisible by 32
{
	static __shared__ sometype shared[32];			// one needs to additionally synchronize threads after execution
	int lane = threadIdx.x % warpSize;			// in the case of multiple use of blockReduceSum in a single kernel
	int wid = threadIdx.x / warpSize;
	val = warpReduceSum(val);
	if (lane==0) shared[wid]=val;
	__syncthreads();
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
	if (wid==0) val = warpReduceSum(val);
	return val;
}
#else
template <class sometype> __inline__ __device__ sometype blockReduceSum(sometype val)	// blockDim.x must be a power of 2
{
	static __shared__ sometype shared[Nthreads];
	shared[threadIdx.x] = val;
	for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
		__syncthreads(); if (threadIdx.x < stride)  shared[threadIdx.x] += shared[threadIdx.x+stride];
	}
	__syncthreads(); return shared[0];
}
#endif

#if (__CUDACC_VER_MAJOR__ < 8) || ( defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600 )
	__device__ double atomicAdd(double* address, double val) // allows to use atomicAdd operation for double precision floating point values
	{ 
		unsigned long long int* address_as_ull = (unsigned long long int*)address; 
		unsigned long long int old = *address_as_ull, assumed; 
		do { 
			assumed = old; 
			old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed))); 
		} while (assumed != old); 
		return __longlong_as_double(old); 
	}
#endif

#if  (defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 320 && MSbits == 64)
	__device__ unsigned long long int atomicXor(unsigned long long int* address, unsigned long long int val) // allows to use atomicXor operation for 64-bit integers
	{
		unsigned int val1,val2; val1 = val & 0xFFFFFFFF; val2 = val>>32;
		val1 = atomicXor((unsigned int*)address,val1);
		val2 = atomicXor((unsigned int*)address+1,val2);
		return (unsigned long long int)val1 + ((unsigned long long int)val2)<<32;
	}

#endif

__global__ void ReplicaInit(Replica* Rd, int rg, int R, unsigned long long rng_seed, unsigned long long initial_sequence){ // initialization of spin lattices of all replicas
	unsigned int B = blockIdx.x, t = threadIdx.x;
	RNGState localrng; hiprand_init(rng_seed,initial_sequence+(t+B*EQthreads),0,&localrng);
	for (unsigned int idx = t; idx < (N/2); idx += EQthreads){
			#if   MSbits == 8
				Rd[B].gA[idx] = hiprand(&localrng) & 0xFF;
				Rd[B].gB[idx] = hiprand(&localrng) & 0xFF;
			#elif MSbits == 16
				Rd[B].gA[idx] = hiprand(&localrng) & 0xFFFF;
				Rd[B].gB[idx] = hiprand(&localrng) & 0xFFFF;
			#elif MSbits == 32
				Rd[B].gA[idx] = hiprand(&localrng);
				Rd[B].gB[idx] = hiprand(&localrng);
			#elif MSbits == 64
				Rd[B].gA[idx] = (((unsigned long long int)hiprand(&localrng))<<32) + hiprand(&localrng) ;
				Rd[B].gB[idx] = (((unsigned long long int)hiprand(&localrng))<<32) + hiprand(&localrng) ;
			#endif
	}
	if(t < MSbits) if((B*MSbits+t)<R) Rd[B].isActive[t] = true; else Rd[B].isActive[t] = false;
}

// parallel spin update
__global__ void checkKerALL(Replica* Rd, int rg, unsigned int sweeps, unsigned long long rng_seed, unsigned long long initial_sequence) // equilibration process
{
	MultiSpin mspin; unsigned int B = blockIdx.x, t = threadIdx.x, ran, idx, i1, i3, i4, tx, ty; // B is replica index

	RNGState localrng; hiprand_init(rng_seed,initial_sequence+(t+blockIdx.x*EQthreads),0,&localrng);

	for(int sweep=0; sweep<sweeps; sweep++){ // sweeps loop

		// sublattice A

		for (idx = t; idx < (N/2); idx += EQthreads){	// sublattice A
			ty = idx / Ldiv2; tx = idx - ty * Ldiv2;
			i1 = ty * Ldiv2 + ((ty&1) ? (tx + 1) : (tx + Ldiv2 - 1)) % Ldiv2;
			i3 = ((ty + L - 1) % L) * Ldiv2 + tx; i4 = ((ty + 1) % L) * Ldiv2 + tx;
			mspin = Rd[B].gA[idx];
			// detecting anti-parallel orientations with NN (Ii = S ^ Ni)
			MultiSpin I1 = mspin ^ Rd[B].gB[i1]; // left- or right-neighbour in B
			MultiSpin I2 = mspin ^ Rd[B].gB[idx]; // right- or left-neighbour spins in the sublattice B
			MultiSpin I3 = mspin ^ Rd[B].gB[i3]; // lower-neighbour spins in the sublattice B
			MultiSpin I4 = mspin ^ Rd[B].gB[i4]; // upper-neighbour spins in the sublattice B
			// performing summation of anti-parallel couplings
			MultiSpin x12 = I1 ^ I2;
			MultiSpin x34 = I3 ^ I4;
			MultiSpin a12 = I1 & I2;
			MultiSpin a34 = I3 & I4;	
			MultiSpin sum0 = x12 ^ x34;
			MultiSpin sum1 = x12 & x34 ^ a12 ^ a34;
			MultiSpin sum2 = a12 & a34;	
			MultiSpin cond4 = 0;
			MultiSpin cond8 = 0; MultiSpin imask=0x1; ran = hiprand(&localrng);
			for (unsigned char i = 0; i < MSbits; ++i){
				cond4 |= (-(ran < tex1Dfetch(boltzT, 0))) & imask;
				cond8 |= (-(ran < tex1Dfetch(boltzT, 1))) & imask;
				imask <<= 1;	ran = AA * ran + CC;
			}
			// acceptance mask
			MultiSpin Acc = (sum1|sum2) | ( (~(sum1|sum2)) & ((sum0&cond4) | (~sum0&cond8)) );
			// Metropolis update + store new configuration to global memory
			Rd[B].gA[idx] = mspin ^ Acc;
		}

		__syncthreads();

		// sublattice B

		for (idx = t; idx < (N/2); idx += EQthreads){	// sublattice B
			ty = idx / Ldiv2; tx = idx - ty * Ldiv2;
			i1 = ty * Ldiv2 + ((ty&1) ? (tx + Ldiv2 - 1) : (tx + 1)) % Ldiv2;
			i3 = ((ty + L - 1) % L) * Ldiv2 + tx; i4 = ((ty + 1) % L) * Ldiv2 + tx;
			mspin = Rd[B].gB[idx];
			MultiSpin I1 = mspin ^ Rd[B].gA[i1]; // left- or right-neighbour in A
			MultiSpin I2 = mspin ^ Rd[B].gA[idx];// right- or left-neighbour spins in the sublattice A
			MultiSpin I3 = mspin ^ Rd[B].gA[i3]; // lower-neighbour spins in the sublattice A
			MultiSpin I4 = mspin ^ Rd[B].gA[i4]; // upper-neighbour spins in the sublattice A
			MultiSpin x12 = I1 ^ I2;
			MultiSpin x34 = I3 ^ I4;
			MultiSpin a12 = I1 & I2;
			MultiSpin a34 = I3 & I4;	
			MultiSpin sum0 = x12 ^ x34;
			MultiSpin sum1 = x12 & x34 ^ a12 ^ a34;
			MultiSpin sum2 = a12 & a34;
			MultiSpin cond4 = 0;
			MultiSpin cond8 = 0; MultiSpin imask=0x1; ran = hiprand(&localrng);
			for (unsigned char i = 0; i < MSbits; ++i){
				cond4 |= (-(ran < tex1Dfetch(boltzT, 0))) & imask;
				cond8 |= (-(ran < tex1Dfetch(boltzT, 1))) & imask;
				imask <<= 1;	ran = AA * ran + CC;
			}	
			MultiSpin Acc = (sum1|sum2) | ( (~(sum1|sum2)) & ((sum0&cond4) | (~sum0&cond8)) );
			Rd[B].gB[idx] = mspin ^ Acc;
		}

		__syncthreads();
	}

}

__global__ void energyKer(Replica* Rd) // calculation of energy and magnetization for each replica
{
	int e, m; unsigned int t = threadIdx.x, idx, iL, iU, B = blockIdx.x, tx, ty;
	MultiSpin sum0, sum1, sum2, sA, sB, Ai2, Bi2, Ai4, Bi4;
	for (idx = t; idx < (N/2); idx += EQthreads){
		if(t < EQthreads){
			sA = Rd[B].gA[idx]; sB = Rd[B].gB[idx];
			ty = idx / Ldiv2; tx = idx - ty * Ldiv2;
			iL = ty * Ldiv2 + (tx + Ldiv2 - 1) % Ldiv2;
			iU = ((ty + 1) % L) * Ldiv2 + tx;
			if(ty&1){ Ai2 = sB; Bi2 = Rd[B].gA[iL];  }
			else{     Ai2 = Rd[B].gB[iL];  Bi2 = sA; }
			Ai4 = Rd[B].gB[iU]; Bi4 = Rd[B].gA[iU];
			// detecting anti-parallel orientations
			MultiSpin I1 = sA ^ Ai2;
			MultiSpin I2 = sA ^ Ai4;
			MultiSpin I3 = sB ^ Bi2;
			MultiSpin I4 = sB ^ Bi4;
			// performing summation of anti-parallel couplings
			MultiSpin x12 = I1 ^ I2;
			MultiSpin x34 = I3 ^ I4;
			MultiSpin a12 = I1 & I2;
			MultiSpin a34 = I3 & I4;
			sum0 = x12 ^ x34;
			sum1 = x12 & x34 ^ a12 ^ a34;
			sum2 = a12 & a34;
		}
		// calculating energy contributions for replicas
		for (unsigned char i = 0; i < MSbits; ++i){
			if(t < EQthreads){
				e = 2*((int)(sum0&0x1) + 2*(int)(sum1&0x1) + 4*(int)(sum2&0x1)) - 4;
				m = 2*((int)(sA&0x1) + (int)(sB&0x1)) - 2;
			} else e = m = 0;
			e = blockReduceSum<int>(e); __syncthreads();
			m = blockReduceSum<int>(m); __syncthreads();
			if (t == 0){
				if (idx==t){
					Rd[B].IE[i] = e;
					Rd[B].M[i]  = m; 
				}else{
					Rd[B].IE[i] += e;
					Rd[B].M[i]  += m; 
				}
			}
			// bit shift operation => moving to next replica in bit string
			sum0 >>= 1;	sum1 >>= 1; sum2 >>= 1;
			sA >>= 1; sB >>= 1;
		}
	}
}

__global__ void QKer(Replica* Rd, int rg, double dB, double Emean, int CalcPart, double* Qd) // calculation of partition function ratio
{
	if(CalcPart==0){			// first part of the calculation
		double factor; int idx = blockIdx.x; int br = threadIdx.x;	// summation of exponential
		factor = Rd[idx].isActive[br] ? exp(-dB*(Rd[idx].IE[br]-Emean)) : 0.0 ;	// Boltzmann-like factors
		#if MSbits < 32
			factor = smallblockReduceSum<double>(factor);
		#else
			factor = blockReduceSum<double>(factor);
		#endif
		if (br == 0) Rd[idx].parSum.ValDouble[0] = factor;	// is saved to global memory
	} else if(CalcPart==1){			// second part of the calculation
		double factor; int t = threadIdx.x; int b = blockIdx.x;
		int idx = t + Nthreads * b;
		factor = (idx < rg) ? Rd[idx].parSum.ValDouble[0]: 0.0;
		factor = blockReduceSum<double>(factor);
		if(t == 0 )  Rd[idx].parSum.ValDouble[1] = factor; // sum for all threads in current block is saved to global memory
	} else{					// third part of the calculation, summation of the partial sums
		double factor; int j, t = threadIdx.x; double MyParSum = 0;
		for (j=0; j<rg; j+=Nthreads){
			factor = (t+j)*Nthreads < rg ? Rd[(t+j)*Nthreads].parSum.ValDouble[1] : 0.0;
			factor = blockReduceSum<double>(factor); __syncthreads();
			MyParSum += factor;
		}
		if(t==0) *Qd = MyParSum;
	}
}

__global__ void CalcTauKer(Replica* Rd, int Rinit, int R, int rg, double lnQ, double dB, unsigned long long rng_seed, unsigned long long initial_sequence) // calculation of numbers of copies for all replicas
{
	int t = threadIdx.x; int b = blockIdx.x;
	unsigned char br = blockIdx.y;			// multispin replica index
	int idx = t + Nthreads * b; double mu, mufloor;
	if (idx < rg) if (Rd[idx].isActive[br]){	// nearest integer resampling
		mu = ((double)Rinit)/R*exp(-dB*(double)Rd[idx].IE[br] - lnQ);
		mufloor = floor(mu);
		RNGState localrng; hiprand_init(rng_seed,initial_sequence+(br+MSbits*idx),0,&localrng);
		if(hiprand_uniform(&localrng) < (mu-mufloor))
			Rd[idx].Roff[br] = mufloor + 1;
		else    Rd[idx].Roff[br] = mufloor;	// number of copies 
	} else Rd[idx].Roff[br] = 0;
}

__global__ void CalcParSum(Replica* Rd, int rg, int CalcPart, int* Rnew)
{
	if(CalcPart==0){	// first part of the calculation
		unsigned int parS; int t = threadIdx.x; int b = blockIdx.x;
		parS = Rd[b].Roff[t]; // (Rd[b].Roff[0] + Rd[b].Roff[1] + ... + Rd[b].Roff[MSbits-1]) is saved to global memory
		#if MSbits < 32
			parS = smallblockReduceSum<unsigned int>(parS);
		#else
			parS = blockReduceSum<unsigned int>(parS);
		#endif
		if(t==0) Rd[b].parSum.ValInt[MSbits] = parS;
	} else if(CalcPart==1){	// second part of the calculation
		unsigned int parS; int t = threadIdx.x; int b = blockIdx.x; int idx = t + b*Nthreads;
		parS = (idx < rg) ? Rd[idx].parSum.ValInt[MSbits] : 0;
		parS = blockReduceSum<unsigned int>(parS);
		// sum of partial sums for replica groups b*Nthreads,b*Nthreads+1,...,(b+1)*Nthreads-1 is saved to global memory.
		if(t==0) Rd[idx].parSum.ValInt[MSbits+1] = parS;
	} else{			// third part of the calculation
		unsigned int parS; int j, t = threadIdx.x, b = blockIdx.x;
		unsigned char br = blockIdx.y; __shared__ unsigned int val;
		int idx = t + Nthreads * b; unsigned int MyParSum = 0;
		for (j = 0; j<b; j+=Nthreads){		// we sum of Roff for all blocks from 0 to (b-1) and for all multi-spin indices.
			parS = (t+j < b) ? Rd[(t+j)*Nthreads].parSum.ValInt[MSbits+1] : 0;
			parS = blockReduceSum<unsigned int>(parS);
			if(t==0) val = parS; __syncthreads(); MyParSum += val;
		}
		if(idx < rg){
			for(j=Nthreads*b;j<idx;j++) MyParSum+=Rd[j].parSum.ValInt[MSbits]; // we add parSum[MSbits] for current block threads from 0 to (t-1)
			for(j=0;j<br;j++) MyParSum+=Rd[idx].Roff[j]; // we add Roff for j = 0,1,..., br-1.
			Rd[idx].parSum.ValInt[br] = MyParSum;                                   // we save partial sum
			if(idx==(rg-1)) if(br==(MSbits-1)) *Rnew = MyParSum + Rd[idx].Roff[br]; // we save new population size
		}
	}
}

__global__ void resampleKer(Replica* Rd, Replica* RdNew, int rg) // renumeration and copying of the replicas (the main part of the resampling process)
{	
	int t = threadIdx.x + blockIdx.z*blockDim.x;	// index of spin variable (from 0 -> N/2-1) 
	int bx = blockIdx.x;				// represents index of group of replicas (j)
	signed char by = blockIdx.y;			// represents index of replica in group/word (k)
	int it_k, it_j;
	#if   MSbits == 64
		unsigned long long int mask = 0x1; mask <<= by;	// mask for selecting spin from old population
		unsigned long long int copy_sourceA = mask & Rd[bx].gA[t];	// selected spin from sublattice A
		unsigned long long int copy_sourceB = mask & Rd[bx].gB[t];	// and B
	#else
		unsigned int mask = 0x1; mask <<= by;			// mask for selecting spin from old population
		unsigned int copy_sourceA = mask & Rd[bx].gA[t];	// selected spin from sublattice A
		unsigned int copy_sourceB = mask & Rd[bx].gB[t];	// and B
	#endif
	for (int p = 0; p < Rd[bx].Roff[by]; ++p){
		it_k = (Rd[bx].parSum.ValInt[by] + p) / rg;
		it_j = (Rd[bx].parSum.ValInt[by] + p) % rg;
		#if   MSbits == 8
			mask = 0x1; mask <<= (it_k + ((t&3)<<3));
			if(copy_sourceA!=0) atomicXor((unsigned int*)&(RdNew[it_j].gA[t-(t&3)]),mask);
			if(copy_sourceB!=0) atomicXor((unsigned int*)&(RdNew[it_j].gB[t-(t&3)]),mask);
		#elif MSbits == 16
			mask = 0x1; mask <<= (it_k + ((t&1)<<4));
			if(copy_sourceA!=0) atomicXor((unsigned int*)&(RdNew[it_j].gA[t-(t&1)]),mask);
			if(copy_sourceB!=0) atomicXor((unsigned int*)&(RdNew[it_j].gB[t-(t&1)]),mask);
		#elif MSbits == 32
			mask = 0x1; mask <<= it_k;
			if(copy_sourceA!=0) atomicXor((unsigned int*)&(RdNew[it_j].gA[t]),mask);
			if(copy_sourceB!=0) atomicXor((unsigned int*)&(RdNew[it_j].gB[t]),mask);
		#elif MSbits == 64
			mask = 0x1; mask <<= it_k;
			if(copy_sourceA!=0) atomicXor((unsigned long long int*)&(RdNew[it_j].gA[t]),mask);
			if(copy_sourceB!=0) atomicXor((unsigned long long int*)&(RdNew[it_j].gB[t]),mask);
		#endif
		if(t==0) 	RdNew[it_j].isActive[it_k] = true;
		else if(t==1)	RdNew[it_j].IE[it_k] = Rd[bx].IE[by];
	}	
}

__global__ void CalcAverages(Replica* Repd, int rg, double* Averages) // calculation of observables via averaging over the population
{
	int t = threadIdx.x, b = blockIdx.x, by = blockIdx.y; int idx = t + Nthreads * b;
	double currE,currE2,currM,currM2,currM4;
	if(idx<rg) if(Repd[idx].isActive[by]){
		currE = Repd[idx].IE[by]; currM = Repd[idx].M[by]; if(currM<0) currM=-currM;
	} else{ currE = 0; currM = 0;} else{ currE = 0; currM = 0;}
	currE2 = currE*currE; currM2 = currM*currM; currM4 = currM2*currM2;
	currE  = blockReduceSum<double>(currE);	 if(t==0) atomicAdd(&Averages[0], currE);  __syncthreads();
	currE2 = blockReduceSum<double>(currE2); if(t==0) atomicAdd(&Averages[1], currE2); __syncthreads();
	currM  = blockReduceSum<double>(currM);	 if(t==0) atomicAdd(&Averages[2], currM);  __syncthreads();
	currM2 = blockReduceSum<double>(currM2); if(t==0) atomicAdd(&Averages[3], currM2); __syncthreads();
	currM4 = blockReduceSum<double>(currM4); if(t==0) atomicAdd(&Averages[4], currM4);
}

#ifdef MHR

__global__ void UpdateShistE(Replica* Repd, int rg, int* ShistE) // adding energy histogram of the current temperature step for the MHR analysis
{
	int t = threadIdx.x, b = blockIdx.x, by = blockIdx.y; int idx = t + Nthreads * b;
	if(idx<rg) if(Repd[idx].isActive[by]){
		atomicAdd(&ShistE[(2*N+Repd[idx].IE[by])/4],1);
	}
}

#endif

#ifdef AdaptiveStep

__global__ void HistogramOverlap(Replica* Repd, int Rinit, int R, int rg, double lnQ, double dB, double* overlap) // calculating histogram overlap
{
	double PartialOverlap;
	int t = threadIdx.x, idx = threadIdx.x + Nthreads * blockIdx.x, by = blockIdx.y;
	if(idx<rg && Repd[idx].isActive[by]) 
		PartialOverlap = min(1.0,((double)Rinit)/R*exp(-dB*(double)Repd[idx].IE[by] - lnQ));
	else PartialOverlap = 0;
	PartialOverlap = blockReduceSum<double>(PartialOverlap);
	if(t==0) atomicAdd(overlap,PartialOverlap);
}

double CalcOverlap(Replica* Rep_d, double dB, int R, double Emean){	// Calculates histogram overlap
		double q, lnQ, ioverlaph;
		int rg = (int)ceil(R/(float)MSbits); 
		int NblocksR = (int)ceil(rg/(double)Nthreads);
		dim3 DimGridR(NblocksR,MSbits,1);
		QKer <<< rg, MSbits >>> (Rep_d, rg, dB, Emean, 0, Qd);
		CUDAErrChk( hipPeekAtLastError() );
		CUDAErrChk( hipDeviceSynchronize() );
		QKer <<< NblocksR, Nthreads >>> (Rep_d, rg, dB, Emean, 1, Qd);
		CUDAErrChk( hipPeekAtLastError() );
		CUDAErrChk( hipDeviceSynchronize() );
		QKer <<< 1, Nthreads >>> (Rep_d, rg, dB, Emean, 2, Qd);
		CUDAErrChk( hipPeekAtLastError() );
		CUDAErrChk( hipDeviceSynchronize() );
		CUDAErrChk( hipMemcpy(&q,Qd,sizeof(double),hipMemcpyDeviceToHost) );
		lnQ = -dB * Emean + log(q) - log((double)R);
		CUDAErrChk( hipMemset(ioverlapd, 0, sizeof(double)) );
		HistogramOverlap<<<DimGridR,Nthreads>>>(Rep_d, Rinit, R, rg, lnQ, dB, ioverlapd);
		CUDAErrChk( hipPeekAtLastError() );
		CUDAErrChk( hipDeviceSynchronize() );
		CUDAErrChk( hipMemcpy(&ioverlaph,ioverlapd,sizeof(double),hipMemcpyDeviceToHost) );
		return (double)ioverlaph/R;
}

#endif

char *optarg; int opterr = 1, optind = 1, optopt, optreset;

int getopt(int nargc, char * const nargv[], const char *ostr)
{
	static char *place = (char*)""; const char *oli;
	if (optreset || !*place) { 
		optreset = 0;
		if (optind >= nargc || *(place = nargv[optind]) != '-') { place = (char*)""; return (-1); }
		if (place[1] && *++place == '-') { ++optind; place = (char*)""; return (-1); }
	}
	if ((optopt = (int)*place++) == (int)':' || !(oli = strchr(ostr, optopt))) {
		if (optopt == (int)'-') return (-1);
		if (!*place) ++optind;
		if (opterr && *ostr != ':') (void)printf("illegal option -- %c\n", optopt);
		return ((int)'?');
	}
	if (*++oli != ':') { optarg = NULL; if (!*place) ++optind; }
	else {
		if (*place) optarg = place; else if (nargc <= ++optind) { 
			place = (char*)""; if (*ostr == ':') return ((int)':');
			if (opterr) (void)printf("option requires an argument -- %c\n", optopt);
			return ((int)'?');
		}
		else optarg = nargv[optind];
		place = (char*)"";  ++optind;
	}
	return (optopt);
}

void PrintParameterUsage(){
	   cout << " Usage: PAisingMSC [options]\n"
		<< " Note: all of the options are optional. Default parameter values are listed in the head of the source code. \n"
		<< " Possible command line options are:\n\n"
		<< " -R Rinit           ( Rinit           = initial size of population of replicas )\n"
		<< " -t EQsweeps        ( EQsweeps        = number of equilibration sweeps )\n"
		<< " -d dBinit          ( dBinit          = inverse temperature step )\n"
		<< " -f Bfin            ( Bfin            = final value of inverse temperature )\n"
		<< " -M runs            ( runs            = number of population annealing algorithm independent runs )\n"
		<< " -s RNGseed         ( RNGseed         = seed for random number generation )\n"
		<< " -P OutputPrecision ( OutputPrecision = precision (number of digits) of the output )\n"
		<< " -o dataDirectory   ( dataDirectory   = data directory name )\n";
}

int main(int argc, char** argv)
{
	// data directory name + create
	char dataDir[200]; unsigned long long rng_seed = RNGseed; int optdir = 0;

	int optc, opti; double optf;
	while ((optc = getopt (argc, argv, "R:t:d:f:M:s:P:o:?")) != -1)	// Processing optional command line options
		switch (optc)
		{
			case 'R': opti = atoi(optarg); if(opti) Rinit = opti; break;           		// -R Rinit
			case 't': opti = atoi(optarg); EQsweeps = opti; break;                 		// -t EQsweeps
			case 'd': optf = atof(optarg); if(optf > 0.0) dBinit = optf; break;     	// -d dBinit
			case 'f': optf = atof(optarg); if(optf > 0.0) Bfin = optf; break;       	// -f Bfin
			case 'M': opti = atoi(optarg); if(opti) runs = opti; break;             	// -M runs
			case 's': opti = atoi(optarg); if(opti) rng_seed = opti; break;         	// -s RNGseed
			case 'P': opti = atoi(optarg); if(opti) OutputPrecision = opti; break;   	// -P OutputPrecision
			case 'o': if(optarg[strlen(optarg)-1]=='/') sprintf(dataDir,"%s",optarg);	// -o dataDir
				  else sprintf(dataDir,"%s/",optarg); optdir = 1; break;
			case '?': PrintParameterUsage();  return 1;
		}
	if(optind < argc){
		for (opti = optind; opti < argc; opti++) fprintf(stderr,"Non-option argument %s\n", argv[opti]);
		return 1;
	}

	#ifdef AdaptiveStep
		if(!optdir) sprintf(dataDir, "./dataMSC_L%d_R%d_EqSw%d/", L, Rinit, EQsweeps);
	#else
		if(!optdir) sprintf(dataDir, "./dataMSC_L%d_R%d_EqSw%d_dB%f/", L, Rinit, EQsweeps, dBinit);
	#endif

	#if defined(_WIN32)
		_mkdir(dataDir);
	#else 
		mkdir(dataDir, 0777);
	#endif
	
	int rmin=0, rmax=runs-1; unsigned long long initial_sequence = 0; int rg;

	double B[nBmax], Binc[nBmax]; B[0]=Binc[0]=Binit; double totPop=0;

	// creating data arrays for thermodynamic variables and errors
	double E[nBmax]; double M[nBmax]; double M2[nBmax]; double M4[nBmax];
	double C[nBmax];
	double lnQ[nBmax]; 			// partition function ratio
	double S[nBmax]; 			// entropy
	double BF[nBmax]; 			// dimensionless free energy estimate
	BF[0] = - N*log(2.0);			// its value at infinite temperature
	int R[nBmax];				// population size
	int nB;

	// CUDAErrChk( hipSetDevice(0) );  // uncomment to explicitly select device number in a setup with multiple cards
	CUDAErrChk(hipDeviceSetCacheConfig(hipFuncCachePreferL1)); // prefer larger L1 cache and smaller shared memory
	// GPU execution time
	hipEvent_t start, stop; float Etime;
	CUDAErrChk( hipEventCreate(&start) );
	CUDAErrChk( hipEventCreate(&stop) );
	// start evaluation time measurement
	hipEventRecord(start, 0);

	double *Averages; double Averages_h[5]; int* Ridev;
	CUDAErrChk( hipMalloc((void**)&Averages,5*sizeof(double)) );
	CUDAErrChk( hipMalloc((void**)&Qd,sizeof(double)) );
	CUDAErrChk( hipMalloc((void**)&Ridev,sizeof(int)) );
	CUDAErrChk( hipMalloc((void**)&ioverlapd,sizeof(double)) );

	// random seed
	cout <<"RNG initial seed: "<< rng_seed<<"\n";
	
	R[0] = Rinit;
	cout << "Memory use of one replica: " << sizeof(Replica) / 1024.0 / (double)MSbits << " kB \n";
	cout << "Memory use of the entire population of " << R[0] << " replicas: "
		<< ceil(R[0]/(double)MSbits)*sizeof(Replica) / 1024.0 / 1024.0 << " MB \n"; fflush(stdout);
	
	// creating energy spectrum for multi-histogram reweighting
	#ifdef MHR
		int Ei[N+1];
		for (int i = 0; i < N+1; ++i){
			Ei[i] = 4*i - 2*N;
		}
	#endif
	
	Replica* Rep_d;
	
	unsigned int boltzGPU[boltzTableL]; // Boltzman factor table - host version
	unsigned int* boltztext;
	
	// memory allocation for Boltzmann factor table
	CUDAErrChk( hipMalloc((void **)&boltztext, boltzTableL * sizeof(unsigned int)) );
	// binding references (global & texture memory buffers)
	CUDAErrChk( hipBindTexture(NULL,boltzT,boltztext,boltzTableL * sizeof(unsigned int)) );
	
	int Ethreads = 1; while(Ethreads < EQthreads) Ethreads <<= 1;

	for (int r = rmin; r <= rmax; ++r){	
		
		rg = (int)ceil(R[0]/(float)MSbits);		// number of replica groups (R / MSbits)
		double sumlnQ = 0.0; double q; double Emean = 0.0;
		CUDAErrChk( hipMalloc((void **)&Rep_d,rg*sizeof(Replica)) );
		int NblocksR = (int)ceil(rg/(float)Nthreads);

		ReplicaInit <<< rg, EQthreads >>> (Rep_d,rg,R[0],rng_seed,initial_sequence); initial_sequence+=rg*EQthreads;
		CUDAErrChk( hipPeekAtLastError() );
		CUDAErrChk( hipDeviceSynchronize() );
		
		// compute energy of all replicas at zero temperature (for 1st resampling)
		energyKer <<< rg, Ethreads >>> (Rep_d);
		CUDAErrChk( hipPeekAtLastError() );
		CUDAErrChk( hipDeviceSynchronize() );
		
		// array for summing the energy histograms over inverse temperatures
		#ifdef MHR
			int ShistE[N+1]; int* ShistEd;
			CUDAErrChk( hipMalloc((void**)&ShistEd,(N+1)*sizeof(int)) );
			CUDAErrChk( hipMemset(ShistEd,0,(N+1)*sizeof(int)) );
			dim3 DimGridR(NblocksR,MSbits,1);
			UpdateShistE<<<DimGridR,Nthreads>>> (Rep_d, rg, ShistEd);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
		#endif
		
		// ------------------------------------------------------------------
		// population annealing
		// ------------------------------------------------------------------
		int i=1, iprev=0; double deltaBeta=dBinit; B[i]=Binc[i]=B[iprev]+deltaBeta;
		
		while(B[i]<=Bfin) {
			// Boltzmann factor tabulation (only two are relevant: exp(-4*B);exp(-8*B))
			boltzGPU[0] = ceil(4294967296.*exp(-4*B[i]));
			boltzGPU[1] = ceil(4294967296.*exp(-8*B[i]));

			// copying table to texture memory - boltztext is bounded with boltzT 
			CUDAErrChk( hipMemcpy(boltztext, boltzGPU, boltzTableL * sizeof(unsigned int),hipMemcpyHostToDevice) );
			
			// compute the partition function ratio - Q
			NblocksR = (int)ceil(rg/(float)Nthreads);
			
			dim3 DimGridR(NblocksR,MSbits,1);

			QKer <<< rg, MSbits >>> (Rep_d, rg, B[i] - B[i-1], Emean, 0, Qd);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
			QKer <<< NblocksR, Nthreads >>> (Rep_d, rg, B[i] - B[i-1], Emean, 1, Qd);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
			QKer <<< 1, Nthreads >>> (Rep_d, rg, B[i] - B[i-1], Emean, 2, Qd);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
			CUDAErrChk( hipMemcpy(&q,Qd,sizeof(double),hipMemcpyDeviceToHost) );

			lnQ[i] = -(B[i] - B[i-1])*Emean + log(q) -log((double)R[i-1]);
			
			CalcTauKer <<< DimGridR, Nthreads >>> (Rep_d, Rinit, R[i-1], rg, lnQ[i], B[i] - B[i-1],rng_seed,initial_sequence); initial_sequence+=rg*MSbits;
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );

			// resampling new population
			CalcParSum <<< rg,    MSbits   >>> (Rep_d, rg, 0, Ridev);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );

			CalcParSum <<< NblocksR, Nthreads >>> (Rep_d, rg, 1, Ridev);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );

			CalcParSum <<< DimGridR, Nthreads >>> (Rep_d, rg, 2, Ridev);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
			CUDAErrChk( hipMemcpy(&R[i], Ridev, sizeof(int),hipMemcpyDeviceToHost) );

			dim3 DimGridRes(rg,MSbits,N/2/EQthreads);        // resampleKer configuration with old value of rg
			rg = (int)ceil(R[i]/(float)MSbits);		// updated number of replica groups
			Replica* RepNew_d;
			CUDAErrChk( hipMalloc((void**)&RepNew_d,rg*sizeof(Replica)) );
			CUDAErrChk( hipMemset(RepNew_d,0,rg*sizeof(Replica)) );
			CUDAErrChk( hipDeviceSynchronize() );

			resampleKer <<< DimGridRes, EQthreads >>> (Rep_d, RepNew_d, rg);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
				
			Replica* RepDel = Rep_d;
			Rep_d = RepNew_d;
			CUDAErrChk( hipFree(RepDel) );

			// equilibrate replicas for certain number of sweeps
			checkKerALL <<< rg, EQthreads >>> (Rep_d,rg,EQsweeps,rng_seed,initial_sequence); initial_sequence+=rg*EQthreads;
			CUDAErrChk( hipPeekAtLastError() ); 
			CUDAErrChk( hipDeviceSynchronize() );

			// compute observables (E,M,O,F)
			// compute energy and magnetization of all replicas
			energyKer <<< rg, Ethreads >>> (Rep_d);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );

			// saving results - energies
			#ifdef EnergiesPopStore
				Replica* Rep_h = (Replica*)malloc(rg*sizeof(Replica));
				CUDAErrChk( hipMemcpy(Rep_h, Rep_d, rg*sizeof(Replica),hipMemcpyDeviceToHost) );
				ofstream results;
				char str[100];
				char str2[100];
				strcpy(str, dataDir);
				sprintf(str2,"PA_energies_%d.dat",i);
				strcat(str,str2);
				results.open(str);
				results.precision(OutputPrecision);
				for (int j = 0; j < rg; ++j)
					for (int l = 0; l < MSbits; ++l)
						if(Rep_h[j].isActive[l]) results << Rep_h[j].IE[l] << " ";
				results.close(); free(Rep_h);
			#endif

			#ifdef MHR
				UpdateShistE<<<DimGridR,Nthreads>>>(Rep_d, rg, ShistEd);
				CUDAErrChk( hipPeekAtLastError() );
				CUDAErrChk( hipDeviceSynchronize() );
			#endif

			CUDAErrChk( hipMemset(Averages, 0, 5*sizeof(double)) );
			CalcAverages<<<DimGridR,Nthreads>>>(Rep_d,rg,Averages);
			CUDAErrChk( hipPeekAtLastError() );
			CUDAErrChk( hipDeviceSynchronize() );
			CUDAErrChk( hipMemcpy(Averages_h,Averages,5*sizeof(double),hipMemcpyDeviceToHost) );

			E[i]  = Emean = Averages_h[0] / R[i];
			C[i]  = (Averages_h[1] / R[i] - E[i]*E[i]) * B[i] * B[i];
			M[i]  = Averages_h[2] / R[i];
			M2[i] = Averages_h[3] / R[i];
			M4[i] = Averages_h[4] / R[i];

			// dimensionless free energy
			sumlnQ -= lnQ[i];
			BF[i] = - N*log(2.0) + sumlnQ;
			// entropy
			S[i] = B[i]*E[i] - BF[i];

	                iprev=i; totPop+=R[i]; i++; 

			if(i>=nBmax){
				#ifdef AdaptiveStep
					fprintf(stderr,"Error: number of temperature steps exceeds nBmax=%d.\n Please consider increasing the population size or decreasing the value of MinOverlap or increasing the value of nBmax.\n",nBmax);
				#else
					fprintf(stderr,"Error: number of temperature steps exceeds nBmax=%d.\n Please consider increasing the inverse temperature step or increasing the value of nBmax.\n",nBmax);
				#endif
				return 1;
			}

			if (r==rmin){
				#ifdef AdaptiveStep
					double overlap, dBmin = 0, dBmax = deltaBeta, dBmean;
					while(1){
						overlap = CalcOverlap ( Rep_d, dBmax, R[iprev], Emean );
						if ( (overlap >= MaxOverlap) && (B[iprev] + dBmax < Bfin) ) dBmax *= 1.1; else break;
					}
					if ( overlap >= MinOverlap ) dBmean = dBmax; 
					else while(1){	// obtaining optimal inverse temperature step with the bisection method
						dBmean = 0.5 * (dBmin + dBmax);
						overlap = CalcOverlap ( Rep_d, dBmean, R[iprev], Emean );
						if ( overlap < MinOverlap ) dBmax = dBmean;
						else if ( overlap >= MaxOverlap ) dBmin = dBmean;
						else break;
					}
					if( (B[iprev] < Bfin) && (B[iprev] + dBmean > Bfin) ) deltaBeta = Bfin - B[iprev]; else deltaBeta = dBmean;
				#endif
				B[i] = Binc[i] = B[iprev] + deltaBeta;
			} else B[i]=Binc[i];
		}
		
		CUDAErrChk( hipFree(Rep_d) );
		nB=i;
		
		// saving results
		{
			ofstream results;
			char str[100];
			char str2[100];
			strcpy(str, dataDir);
			sprintf(str2, "PA_results_run_%d.dat", r);
			strcat(str,str2);
			results.open(str);
			results.precision(OutputPrecision);
			for (int i = 0; i < nB; ++i) {
				results << B[i] << " "
					<< E[i] / N << " "
					<< C[i] / N << " "
					<< M[i] / N << " "
					<< M2[i] / N / N << " "
					<< M4[i] / N / N / N / N << " "
					<< BF[i] / N << " "
					<< S[i] / N << " "
					<< R[i] << " "
					<< lnQ[i] << "\n";
			}
			results.close();
		}
		
		// multi-histogam reweighting (MHR) analysis
		#ifdef MHR
			// declaring arrays used in MHR analysis
			double lnOmega[N+1];
			double E_MHR[nB*MHR_Niter];
			double C_MHR[nB*MHR_Niter];
			double BF_MHR[nB*MHR_Niter];

			bool relTerm[N+1];

			CUDAErrChk( hipMemcpy(ShistE,ShistEd,(N+1)*sizeof(int),hipMemcpyDeviceToHost) );

			for (int l = 0; l < MHR_Niter; ++l){
				// calculate lnOmega
				double Sigma[nB];
				double mSigma;
				for (int k = 0; k < N+1; ++k){				
					// maxima of -S = BF - B*E
					Sigma[0] = BF[0]-B[0]*Ei[k];
					mSigma = Sigma[0];
					for (int i = 1; i < nB; ++i){
						Sigma[i] = BF[i]-B[i]*Ei[k];
						if (mSigma < Sigma[i]){
							mSigma = Sigma[i];
						}
					}
					double sD = 0;
					for (int i = 0; i < nB; ++i){
						sD += R[i]*exp(Sigma[i]-mSigma);
					}
					if ((ShistE[k] == 0) || (sD == 0)){
						relTerm[k] = false;
						lnOmega[k] = 0;
					} else {
						relTerm[k] = true;
						lnOmega[k] = log(ShistE[k]) - mSigma - log(sD);
					}
				}
				// reweigting of observables
				double expOm[N+1];
				double Om[N+1];
				double mOm;
				for (int i = 0; i < nB; ++i){
					// determine the maxima of the reweighting exponent
					mOm = lnOmega[0] - B[i]*Ei[0];
					for (int k = 0; k < N+1; ++k){
						Om[k] = lnOmega[k] - B[i]*Ei[k];
						if (mOm < Om[k]){
							mOm = Om[k];
						}
					}
					// calculate reweighting exponentials
					double p = 0;
					for (int k = 0; k < N+1; ++k){
						expOm[k] = exp(Om[k] - mOm);
						if (relTerm[k])
							p += expOm[k];
					}
					double s = 0; 
					for (int k = 0; k < N+1; ++k){
						if (relTerm[k])
							s += Ei[k]*expOm[k];
					}
					E_MHR[i+l*nB] = s / p / N;
					BF_MHR[i+l*nB] = - mOm - log(p);
					BF[i] = BF_MHR[i+l*nB];
					s = 0;
					for (int k = 0; k < N+1; ++k){
						if (relTerm[k])
							s += pow(Ei[k]-E_MHR[i+l*nB]*N,2)*expOm[k];
					}
					C_MHR[i+l*nB] = B[i]*B[i] * s / p / N;
				}
			}
			// saving results
			{
				ofstream results;
				char MHRDataFile[100];
				char str2[100];
				strcpy(MHRDataFile, dataDir);
				sprintf(str2,"PA_MHR_results_run_%d.dat",r);
				strcat(MHRDataFile,str2);
				results.open(MHRDataFile);
				results.precision(OutputPrecision);
				for (int i = 0; i < nB; ++i){
					results << B[i] << " ";
					for (int l = 0; l < MHR_Niter; ++l){
						results << E_MHR[i+l*nB] << " ";
						results << C_MHR[i+l*nB] << " ";
						results << BF_MHR[i+l*nB] / N << " ";
					}
					results << "\n";
				}
				results.close();
			}

			CUDAErrChk( hipFree(ShistEd) );

		#endif
	}

	CUDAErrChk( hipFree(Averages) );
	CUDAErrChk( hipFree(Ridev) );
	CUDAErrChk( hipFree(Qd) );	
	CUDAErrChk( hipFree(ioverlapd) );
	CUDAErrChk( hipUnbindTexture(boltzT) );
	CUDAErrChk( hipFree(boltztext));

	CUDAErrChk( hipDeviceSynchronize() );
	CUDAErrChk( hipEventRecord(stop, 0) );
	CUDAErrChk( hipEventSynchronize(stop) );	
	CUDAErrChk( hipEventElapsedTime(&Etime, start, stop) );
	cout << "Elapsed time: " << setprecision(8) << Etime/1000 << " s\n";
	cout << "Time per spin-flip: " << setprecision(8) << Etime*1e6/EQsweeps/N/totPop << " ns\n";

	CUDAErrChk( hipEventDestroy(start) );
	CUDAErrChk( hipEventDestroy(stop) );

	return 0;
}
